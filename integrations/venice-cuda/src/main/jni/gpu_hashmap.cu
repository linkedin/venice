#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <atomic>

// GPU HashMap structures
struct GPUHashEntry {
    uint64_t key_hash;
    uint32_t key_size;
    uint32_t value_size;
    uint32_t key_offset;    // Offset in data pool
    uint32_t value_offset;  // Offset in data pool
    int32_t next;          // Index of next entry in chain (-1 if end)
    uint32_t flags;        // 0 = empty, 1 = occupied, 2 = deleted
};

struct GPUHashMap {
    GPUHashEntry* entries;
    uint8_t* data_pool;      // Pool for storing actual key/value data
    int32_t* bucket_heads;   // Head indices for each bucket
    uint32_t num_buckets;
    uint32_t max_entries;
    uint32_t data_pool_size;
    uint32_t* entry_count;   // Atomic counter for entries
    uint32_t* data_pool_used; // Atomic counter for data pool usage
    uint32_t* next_entry_idx; // Atomic counter for next available entry slot
};

struct GPUContext {
    int device_id;
    hipStream_t stream;
    bool initialized;
};

// MurmurHash3 for GPU
__device__ uint64_t gpu_murmur3_64(const uint8_t* key, uint32_t len) {
    const uint64_t m = 0xc6a4a7935bd1e995ULL;
    const int r = 47;
    uint64_t h = 0x8445d61a4e774912ULL ^ (len * m);
    
    const uint64_t* data = (const uint64_t*)key;
    const uint64_t* end = data + (len / 8);
    
    while (data != end) {
        uint64_t k = *data++;
        k *= m;
        k ^= k >> r;
        k *= m;
        h ^= k;
        h *= m;
    }
    
    const uint8_t* data2 = (const uint8_t*)data;
    switch (len & 7) {
        case 7: h ^= ((uint64_t)data2[6]) << 48;
        case 6: h ^= ((uint64_t)data2[5]) << 40;
        case 5: h ^= ((uint64_t)data2[4]) << 32;
        case 4: h ^= ((uint64_t)data2[3]) << 24;
        case 3: h ^= ((uint64_t)data2[2]) << 16;
        case 2: h ^= ((uint64_t)data2[1]) << 8;
        case 1: h ^= ((uint64_t)data2[0]);
                h *= m;
    }
    
    h ^= h >> r;
    h *= m;
    h ^= h >> r;
    
    return h;
}

// CUDA kernel for inserting into hashmap
__global__ void gpu_insert_kernel(GPUHashMap* map, const uint8_t* keys, const uint32_t* key_sizes,
                                  const uint8_t* values, const uint32_t* value_sizes,
                                  uint32_t* key_offsets, uint32_t* value_offsets,
                                  bool* results, uint32_t num_inserts) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_inserts) return;
    
    // Calculate offsets for this thread's key and value
    uint32_t key_offset = key_offsets[tid];
    uint32_t value_offset = value_offsets[tid];
    uint32_t key_size = key_sizes[tid];
    uint32_t value_size = value_sizes[tid];
    
    const uint8_t* key = keys + key_offset;
    const uint8_t* value = values + value_offset;
    
    // Hash the key
    uint64_t hash = gpu_murmur3_64(key, key_size);
    uint32_t bucket = hash % map->num_buckets;
    
    // Allocate space in data pool
    uint32_t data_offset = atomicAdd(map->data_pool_used, key_size + value_size);
    if (data_offset + key_size + value_size > map->data_pool_size) {
        results[tid] = false;
        return;
    }
    
    // Copy key and value to data pool
    memcpy(map->data_pool + data_offset, key, key_size);
    memcpy(map->data_pool + data_offset + key_size, value, value_size);
    
    // Allocate entry
    uint32_t entry_idx = atomicAdd(map->next_entry_idx, 1);
    if (entry_idx >= map->max_entries) {
        results[tid] = false;
        return;
    }
    
    // Fill entry
    GPUHashEntry* entry = &map->entries[entry_idx];
    entry->key_hash = hash;
    entry->key_size = key_size;
    entry->value_size = value_size;
    entry->key_offset = data_offset;
    entry->value_offset = data_offset + key_size;
    entry->flags = 1; // occupied
    
    // Insert into bucket chain
    int32_t old_head = atomicExch(&map->bucket_heads[bucket], entry_idx);
    entry->next = old_head;
    
    atomicAdd(map->entry_count, 1);
    results[tid] = true;
}

// CUDA kernel for lookup
__global__ void gpu_lookup_kernel(GPUHashMap* map, const uint8_t* keys, const uint32_t* key_sizes,
                                  uint32_t* key_offsets, uint8_t* values, uint32_t* value_sizes,
                                  uint32_t* value_offsets, bool* found, uint32_t num_lookups) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_lookups) return;
    
    uint32_t key_offset = key_offsets[tid];
    uint32_t key_size = key_sizes[tid];
    const uint8_t* key = keys + key_offset;
    
    // Hash the key
    uint64_t hash = gpu_murmur3_64(key, key_size);
    uint32_t bucket = hash % map->num_buckets;
    
    // Search in bucket chain
    int32_t current = map->bucket_heads[bucket];
    found[tid] = false;
    
    while (current != -1) {
        GPUHashEntry* entry = &map->entries[current];
        
        if (entry->flags == 1 && entry->key_hash == hash && entry->key_size == key_size) {
            // Compare actual keys
            bool match = true;
            for (uint32_t i = 0; i < key_size; i++) {
                if (map->data_pool[entry->key_offset + i] != key[i]) {
                    match = false;
                    break;
                }
            }
            
            if (match) {
                // Found! Copy value
                uint32_t value_offset = value_offsets[tid];
                memcpy(values + value_offset, map->data_pool + entry->value_offset, entry->value_size);
                value_sizes[tid] = entry->value_size;
                found[tid] = true;
                return;
            }
        }
        
        current = entry->next;
    }
}

// Host-side functions
extern "C" {

GPUContext* gpu_initialize(int device_id) {
    GPUContext* ctx = new GPUContext();
    ctx->device_id = device_id;
    ctx->initialized = false;
    
    hipError_t err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        delete ctx;
        return nullptr;
    }
    
    err = hipStreamCreate(&ctx->stream);
    if (err != hipSuccess) {
        delete ctx;
        return nullptr;
    }
    
    ctx->initialized = true;
    return ctx;
}

void gpu_shutdown(GPUContext* ctx) {
    if (ctx && ctx->initialized) {
        hipStreamDestroy(ctx->stream);
        delete ctx;
    }
}

GPUHashMap* gpu_create_hashmap(GPUContext* ctx, uint32_t num_buckets, uint32_t max_entries, uint32_t data_pool_size) {
    if (!ctx || !ctx->initialized) return nullptr;
    
    hipSetDevice(ctx->device_id);
    
    GPUHashMap* h_map = new GPUHashMap();
    h_map->num_buckets = num_buckets;
    h_map->max_entries = max_entries;
    h_map->data_pool_size = data_pool_size;
    
    // Allocate GPU memory
    hipMalloc(&h_map->entries, sizeof(GPUHashEntry) * max_entries);
    hipMalloc(&h_map->data_pool, data_pool_size);
    hipMalloc(&h_map->bucket_heads, sizeof(int32_t) * num_buckets);
    hipMalloc(&h_map->entry_count, sizeof(uint32_t));
    hipMalloc(&h_map->data_pool_used, sizeof(uint32_t));
    hipMalloc(&h_map->next_entry_idx, sizeof(uint32_t));
    
    // Initialize
    hipMemset(h_map->entries, 0, sizeof(GPUHashEntry) * max_entries);
    hipMemset(h_map->bucket_heads, -1, sizeof(int32_t) * num_buckets);
    hipMemset(h_map->entry_count, 0, sizeof(uint32_t));
    hipMemset(h_map->data_pool_used, 0, sizeof(uint32_t));
    hipMemset(h_map->next_entry_idx, 0, sizeof(uint32_t));
    
    // Allocate device copy of hashmap struct
    GPUHashMap* d_map;
    hipMalloc(&d_map, sizeof(GPUHashMap));
    hipMemcpy(d_map, h_map, sizeof(GPUHashMap), hipMemcpyHostToDevice);
    
    // Store device pointer in host struct for later use
    h_map->entries = (GPUHashEntry*)d_map;
    
    return h_map;
}

void gpu_destroy_hashmap(GPUHashMap* map) {
    if (!map) return;
    
    GPUHashMap* d_map = (GPUHashMap*)map->entries;
    GPUHashMap h_map;
    hipMemcpy(&h_map, d_map, sizeof(GPUHashMap), hipMemcpyDeviceToHost);
    
    hipFree(h_map.entries);
    hipFree(h_map.data_pool);
    hipFree(h_map.bucket_heads);
    hipFree(h_map.entry_count);
    hipFree(h_map.data_pool_used);
    hipFree(h_map.next_entry_idx);
    hipFree(d_map);
    
    delete map;
}

bool gpu_insert_batch(GPUContext* ctx, GPUHashMap* map, 
                     const uint8_t* keys, const uint32_t* key_sizes,
                     const uint8_t* values, const uint32_t* value_sizes,
                     uint32_t num_inserts) {
    if (!ctx || !map || !keys || !values || num_inserts == 0) return false;
    
    hipSetDevice(ctx->device_id);
    
    // Calculate offsets
    uint32_t* h_key_offsets = new uint32_t[num_inserts];
    uint32_t* h_value_offsets = new uint32_t[num_inserts];
    uint32_t key_total = 0, value_total = 0;
    
    for (uint32_t i = 0; i < num_inserts; i++) {
        h_key_offsets[i] = key_total;
        h_value_offsets[i] = value_total;
        key_total += key_sizes[i];
        value_total += value_sizes[i];
    }
    
    // Allocate device memory
    uint8_t *d_keys, *d_values;
    uint32_t *d_key_sizes, *d_value_sizes, *d_key_offsets, *d_value_offsets;
    bool *d_results;
    
    hipMalloc(&d_keys, key_total);
    hipMalloc(&d_values, value_total);
    hipMalloc(&d_key_sizes, sizeof(uint32_t) * num_inserts);
    hipMalloc(&d_value_sizes, sizeof(uint32_t) * num_inserts);
    hipMalloc(&d_key_offsets, sizeof(uint32_t) * num_inserts);
    hipMalloc(&d_value_offsets, sizeof(uint32_t) * num_inserts);
    hipMalloc(&d_results, sizeof(bool) * num_inserts);
    
    // Copy data to device
    hipMemcpy(d_keys, keys, key_total, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, value_total, hipMemcpyHostToDevice);
    hipMemcpy(d_key_sizes, key_sizes, sizeof(uint32_t) * num_inserts, hipMemcpyHostToDevice);
    hipMemcpy(d_value_sizes, value_sizes, sizeof(uint32_t) * num_inserts, hipMemcpyHostToDevice);
    hipMemcpy(d_key_offsets, h_key_offsets, sizeof(uint32_t) * num_inserts, hipMemcpyHostToDevice);
    hipMemcpy(d_value_offsets, h_value_offsets, sizeof(uint32_t) * num_inserts, hipMemcpyHostToDevice);
    
    // Launch kernel
    uint32_t threads_per_block = 256;
    uint32_t blocks = (num_inserts + threads_per_block - 1) / threads_per_block;
    
    GPUHashMap* d_map = (GPUHashMap*)map->entries;
    gpu_insert_kernel<<<blocks, threads_per_block, 0, ctx->stream>>>(
        d_map, d_keys, d_key_sizes, d_values, d_value_sizes,
        d_key_offsets, d_value_offsets, d_results, num_inserts
    );
    
    // Wait for completion
    hipStreamSynchronize(ctx->stream);
    
    // Check results
    bool* h_results = new bool[num_inserts];
    hipMemcpy(h_results, d_results, sizeof(bool) * num_inserts, hipMemcpyDeviceToHost);
    
    bool all_success = true;
    for (uint32_t i = 0; i < num_inserts; i++) {
        if (!h_results[i]) {
            all_success = false;
            break;
        }
    }
    
    // Cleanup
    delete[] h_key_offsets;
    delete[] h_value_offsets;
    delete[] h_results;
    
    hipFree(d_keys);
    hipFree(d_values);
    hipFree(d_key_sizes);
    hipFree(d_value_sizes);
    hipFree(d_key_offsets);
    hipFree(d_value_offsets);
    hipFree(d_results);
    
    return all_success;
}

} // extern "C"
